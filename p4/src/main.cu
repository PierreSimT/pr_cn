#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <chrono>
#include <opencv2/opencv.hpp>

using namespace cv;

__global__ void gaussBlurCUDA(uchar * src, uchar * result, size_t filas, size_t columnas, float radius) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    //printf("Procesando fila %d \n", i);

    double rs = ceil(radius * 2.57);
    if ( i < filas ) {
        for (int j = 0; j < columnas; j++) {       
            float val = 0, sum = 0;
            for (int t = i - rs; t < i + rs + 1; t++)
            {
                for (int s = j - rs; s < j + rs + 1; s++)
                {
                    int x = min((int)columnas - 1, max(0, s));
                    int y = min((int)filas - 1, max(0, t));

                    float dsq = (s - j) * (s - j) + (t - i) * (t - i);
                    float weight = exp(-dsq / (2.0 * radius * radius)) / (M_PI * 2.0 * radius * radius);

                    val += src[y * columnas + x] * weight;
                    sum += weight;
                }

                result[i * columnas + j] = round(val / sum);
            }
        }
    }

}

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        printf("uso: a.out <Image_Path> <Gauss_Radius>\n");
        return -1;
    }

    Mat img = imread(argv[1], CV_LOAD_IMAGE_GRAYSCALE);

    Mat *resultCPU = new Mat(img.rows, img.cols, CV_8UC1);

    float radius = atof(argv[2]);
    uchar * src;
    uchar * result;

    // PREPARAR DATOS CUDA
    hipMalloc(&src, img.total() * sizeof(uchar));
    hipMalloc(&result, img.total() * sizeof(uchar));
    hipMemcpy(src, img.data, img.total() * sizeof(uchar), hipMemcpyHostToDevice);
    
    // PREPARAR GRIDS, BLOQUES Y THREADS
    int numBloques = 16;
    dim3 threadsPerBlock(img.rows / numBloques);

    std::cout << "Ejecutando filtro Gaussiano..." << std::endl;

    auto t1 = std::chrono::high_resolution_clock::now();
    gaussBlurCUDA<<<numBloques, threadsPerBlock>>>(src, result, img.rows, img.cols, radius);
    hipMemcpy(resultCPU->data, result, img.total(), hipMemcpyDeviceToHost);
    auto t2 = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
    std::cout << "Tiempo de ejecucion: " << (float) (duration / 1000.0) << " sec" << std::endl;

    imwrite("result.png", *resultCPU);
    std::cout << "Resultado escrito en ./result.png" << std::endl;

    // FINALIZAR CUDA
    hipFree(src);
    hipFree(result);
    
    return 0;
}